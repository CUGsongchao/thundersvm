#include "hip/hip_runtime.h"
//
// Created by jiashuai on 17-9-20.
//
#include <thundersvm/syncdata.h>
#include <hipsparse.h>
#include "thundersvm/kernel/kernelmatrix_kernel.h"
#include <config.h>

#ifdef USE_CUDA
namespace svm_kernel {
    __global__ void
    kernel_get_working_set_ins(const float_type *val, const int *col_ind, const int *row_ptr, const int *data_row_idx,
                               float_type *data_rows,
                               int m) {
        KERNEL_LOOP(i, m) {
            int row = data_row_idx[i];
            for (int j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
                int col = col_ind[j];
                data_rows[col * m + i] = val[j]; // row-major for cuSPARSE
            }
        }
    }

    __global__ void
    kernel_RBF_kernel(const float_type *self_dot0, const float_type *self_dot1, float_type *dot_product, int m, int n,
                      float_type gamma) {
        //m rows of kernel matrix, where m is the working set size; n is the number of training instances
        KERNEL_LOOP(idx, m * n) {
            int i = idx / n;//i is row id
            int j = idx % n;//j is column id
            dot_product[idx] = expf(-(self_dot0[i] + self_dot1[j] - dot_product[idx] * 2) * gamma);
        }
    }

    __global__ void
    kernel_RBF_kernel(const int *self_dot0_idx, const float_type *self_dot1, float_type *dot_product, int m, int n,
                      float_type gamma) {
        //compute m rows of kernel matrix, where m is the working set size and n is the number of training instances, according to idx
        KERNEL_LOOP(idx, m * n) {
            int i = idx / n;//i is row id
            int j = idx % n;//j is column id
            dot_product[idx] = expf(-(self_dot1[self_dot0_idx[i]] + self_dot1[j] - dot_product[idx] * 2) * gamma);
        }
    }

    __global__ void
    kernel_sum_kernel_values(const float_type *coef, int total_sv, const int *sv_start, const int *sv_count,
                             const float_type *rho,
                             const float_type *k_mat, float_type *dec_values, int n_classes, int n_instances) {
        KERNEL_LOOP(idx, n_instances) {
            int k = 0;
            int n_binary_models = n_classes * (n_classes - 1) / 2;
            for (int i = 0; i < n_classes; ++i) {
                for (int j = i + 1; j < n_classes; ++j) {
                    int si = sv_start[i];
                    int sj = sv_start[j];
                    int ci = sv_count[i];
                    int cj = sv_count[j];
                    const float_type *coef1 = &coef[(j - 1) * total_sv];
                    const float_type *coef2 = &coef[i * total_sv];
                    const float_type *k_values = &k_mat[idx * total_sv];
                    float_type sum = 0;
                    for (int l = 0; l < ci; ++l) {
                        sum += coef1[si + l] * k_values[si + l];
                    }
                    for (int l = 0; l < cj; ++l) {
                        sum += coef2[sj + l] * k_values[sj + l];
                    }
                    dec_values[idx * n_binary_models + k] = sum - rho[k];
                    k++;
                }
            }
        }
    }

    __global__ void
    kernel_poly_kernel(float_type *dot_product, float_type gamma, float_type coef0, int degree, int mn) {
        KERNEL_LOOP(idx, mn) {
            dot_product[idx] = powf(gamma * dot_product[idx] + coef0, degree);
        }
    }

    __global__ void kernel_sigmoid_kernel(float_type *dot_product, float_type gamma, float_type coef0, int mn) {
        KERNEL_LOOP(idx, mn) {
            dot_product[idx] = tanhf(gamma * dot_product[idx] + coef0);
        }
    }

    void sum_kernel_values(const SyncData<float_type> &coef, int total_sv, const SyncData<int> &sv_start,
                           const SyncData<int> &sv_count, const SyncData<float_type> &rho,
                           const SyncData<float_type> &k_mat,
                           SyncData<float_type> &dec_values, int n_classes, int n_instances) {
        SAFE_KERNEL_LAUNCH(kernel_sum_kernel_values, coef.device_data(), total_sv, sv_start.device_data(),
                           sv_count.device_data(), rho.device_data(), k_mat.device_data(), dec_values.device_data(),
                           n_classes, n_instances);

    }

    void
    get_working_set_ins(const SyncData<float_type> &val, const SyncData<int> &col_ind, const SyncData<int> &row_ptr,
                        const SyncData<int> &data_row_idx, SyncData<float_type> &data_rows, int m) {
        SAFE_KERNEL_LAUNCH(kernel_get_working_set_ins, val.device_data(), col_ind.device_data(), row_ptr.device_data(),
                           data_row_idx.device_data(), data_rows.device_data(), m);

    }

    void
    RBF_kernel(const SyncData<float_type> &self_dot0, const SyncData<float_type> &self_dot1,
               SyncData<float_type> &dot_product, int m,
               int n,
               float_type gamma) {
        SAFE_KERNEL_LAUNCH(kernel_RBF_kernel, self_dot0.device_data(), self_dot1.device_data(),
                           dot_product.device_data(), m, n, gamma);
    }

    void
    RBF_kernel(const SyncData<int> &self_dot0_idx, const SyncData<float_type> &self_dot1,
               SyncData<float_type> &dot_product, int m,
               int n, float_type gamma) {
        SAFE_KERNEL_LAUNCH(kernel_RBF_kernel, self_dot0_idx.device_data(), self_dot1.device_data(),
                           dot_product.device_data(), m, n, gamma);
    }

    void poly_kernel(SyncData<float_type> &dot_product, float_type gamma, float_type coef0, int degree, int mn) {
        SAFE_KERNEL_LAUNCH(kernel_poly_kernel, dot_product.device_data(), gamma, coef0, degree, mn);
    }

    void sigmoid_kernel(SyncData<float_type> &dot_product, float_type gamma, float_type coef0, int mn) {
        SAFE_KERNEL_LAUNCH(kernel_sigmoid_kernel, dot_product.device_data(), gamma, coef0, mn);
    }

    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    bool cusparse_init;

    void dns_csr_mul(int m, int n, int k, const SyncData<float_type> &dense_mat, const SyncData<float_type> &csr_val,
                     const SyncData<int> &csr_row_ptr, const SyncData<int> &csr_col_ind, int nnz,
                     SyncData<float_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }
        float one(1);
        float zero(0);
        hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                        m, n, k, nnz, &one, descr, csr_val.device_data(), csr_row_ptr.device_data(),
                        csr_col_ind.device_data(),
                        dense_mat.device_data(), n, &zero, result.device_data(), m);
        //hipsparseScsrmm return row-major matrix, so no transpose is needed
    }
}
#endif
